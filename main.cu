#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// 
// 	File name: main.cu
//	Created By: Haard Panchal
//	Create Date: 03/11/2020
//
//	Description:
//		Main file for the Ray Tracing project. The file implements the parallel CUDA algorithm.
//      You can also use it to create the world that would be used to render the final result.
//
//	History:
//		03/10/19: H. Panchal Created the file
//
//  Declaration:
//      N/A
//
// ----------------------------------------------------------------------------------------------------

// #define CUDADEBUG
// #define RENDERDEBUG
#define ACTUALRENDER
// #define INITDEBUG

#include <iostream>
#include <math.h>

#include "Vector3.h"
#include "Ray.h"

#include "Camera.h"
#include "World.h"

#include "Sphere.h"
#include "Plane.h"
#include "PointLight.h"
#include "DirectionalLight.h"
#include "SpotLight.h"

#include "RenderEngine.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*  Function: initializeEngine
//
//  The function adds different objects to World
//
//	Parameters:
//      World ** world: A pointer to a pointer to a world object
//		int w: The width of the resulting image
//		int h: The height of the resulting image
//	
//	Return:
//		void
*/
__global__
void initializeWorld(World ** world, int w, int h) {
    *world = new World();

    Vector3 color(0.0f, 0.5f, 0.0f);
    Vector3 center(-1.0, 0.0, 0.0);
    float r = 1.0f;
    Sphere * s = new Sphere(center, r, color);

    (*world)->addVisibleObject(s);

    float beam_angle = 10.0;
    float falloff_angle = 30.0;
    beam_angle = beam_angle * PI / 180.0;
    falloff_angle = falloff_angle * PI / 180.0;
    Vector3 spotlightpos(-0.3, 0.25, 3.0f);
    Vector3 spotlightdir = -spotlightpos;
    SpotLight * spotlight = new SpotLight(spotlightpos, spotlightdir, beam_angle, falloff_angle);

    (*world)->addLight(spotlight);

    Vector3 color2(0.5f, 1.0f, 0.25f);
    Vector3 point(0.0, -2.5, 0.0);
    Vector3 normal(0, 1.0, 0.0);
    Plane * p = new Plane(normal, point, color2);

    (*world)->addVisibleObject(p);

    Vector3 positioncam(0.0, 0.0, 5.0);
    Vector3 lookat(0.0f, 0.0f, 0.0f);
    Vector3 direction = lookat - positioncam;
    Vector3 updir(0.0, 1.0, 0.0);
    float aspect_ratio = (w * 1.0)/(h * 1.0);
    float distance_from_screen = 1.0;
    Camera * cam = new Camera(positioncam, direction, updir, aspect_ratio, 1.0, distance_from_screen);

    (*world)->setCamera(*cam);
}

/*  Function: addWorldToEngine
//
//	The function initializes the RenderEngine
//  An already initialized World object is passed to the RenderEngine
//
//	Parameters:
//      int w: Width of the rendered image
//      int h: Height of the rendered image		
//		RenderEngine ** r_engine: Pointer to a pointer to the RenderEngine object
//      World ** world: Pointer to a pointer 	
// 
//	Return:
//		void
*/
__global__
void addWorldToEngine(int w, int h, RenderEngine ** r_engine, World ** world) {
    *r_engine = new RenderEngine(w, h, **world);
}



/*  Function: Parallelize Render for each pixels
//
//	The kernel CUDA function implements the parallel threads for rendering each pixel.
//  The rendered pixels are stored in the frame_buffer array
//
//	Parameters:
//
//		
//		
//	
//	Return:
//		void
*/
__global__
void renderPixels(RenderEngine ** r_engine, Vector3 * frame_buffer, int w, int h) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int index_ij = j * w + i;

    frame_buffer[index_ij] =  (*r_engine)->renderPixel(i, j);
    #ifdef CUDADEBUG
    printf("End of renderPixels\n");
    printf("framebuffer: i: %d r: %d c: %d\n", index_ij, i, j);
    #endif
}


/*  Function: main
//
//	Parses the argument list. Initializes the relevant objects and starts rendering.
//
//	Parameters:
//
//		int argc: Number of arguments
//		char *argv[]: List of the arguments
//	
//	Return:
//		int: 0 if successful
*/
int main(int argc, char *argv[]) {

    int wid_cuda = 1200, hgt_cuda = 800;

    Vector3 * frame_buffer_cuda;
    gpuErrchk(hipMallocManaged(&frame_buffer_cuda, wid_cuda * hgt_cuda * sizeof(Vector3)));

    World ** world_cuda;
    gpuErrchk(hipMallocManaged(&world_cuda, sizeof(World *)));

    RenderEngine ** r_engine_cuda;
    gpuErrchk(hipMallocManaged(&r_engine_cuda, sizeof(RenderEngine *)));

    initializeWorld<<<1, 1>>>(world_cuda, wid_cuda, hgt_cuda);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    addWorldToEngine<<<1, 1>>>(wid_cuda, hgt_cuda, r_engine_cuda, world_cuda);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    const int block_size_side = 16;
    const dim3 block_size(block_size_side, block_size_side);
    const int grid_size_hgt = (hgt_cuda + block_size_side - 1)/block_size_side;
    const int grid_size_wid = (wid_cuda + block_size_side - 1)/block_size_side;
    const dim3 grid_size(grid_size_wid, grid_size_hgt);

    #ifdef CUDADEBUG
    std::cout<<"Grid Sizes: "<<grid_size_hgt<<" "<<grid_size_wid<<std::endl;
    std::cout<<"Block Sizes: "<<block_size_side<<" "<<block_size_side<<std::endl;
    #endif

    renderPixels<<<grid_size, block_size>>>(r_engine_cuda, frame_buffer_cuda, wid_cuda, hgt_cuda);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    makeImage(frame_buffer_cuda, wid_cuda, hgt_cuda);

    return 0;
}