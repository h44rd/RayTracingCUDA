#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// 
// 	File name: main.cu
//	Created By: Haard Panchal
//	Create Date: 03/11/2020
//
//	Description:
//		Main file for the Ray Tracing project. The file implements the parallel CUDA algorithm.
//      You can also use it to create the world that would be used to render the final result.
//
//	History:
//		03/10/19: H. Panchal Created the file
//
//  Declaration:
//      N/A
//
// ----------------------------------------------------------------------------------------------------

// #define MATERIALDEBUG
// #define AREALIGHTDEBUG
// #define SHADOWDEBUG
// #define CUDADEBUG
// #define RENDERDEBUG
#define ACTUALRENDER
// #define INITDEBUG

#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb_image.h"

#include "Vector3.h"
#include "Ray.h"

#include "Camera.h"
#include "World.h"

#include "Sphere.h"
#include "Plane.h"
#include "PointLight.h"
#include "DirectionalLight.h"
#include "SpotLight.h"
#include "AreaLight.h"

#include "Material.h"
#include "TextureMaterial.h"

#include "RenderEngine.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/*  Function: initializeEngine
//
//  The function adds different objects to World.
//  All the object must be initialized onto the heap.
//
//	Parameters:
//      World ** world: A pointer to a pointer to a world object
//		int w: The width of the resulting image
//		int h: The height of the resulting image
//	
//	Return:
//		void
*/
__global__
void initializeWorld(World ** world, int w, int h, unsigned char ** array_of_images, int * img_w, int * img_h, int * img_chns, int n_imgs) {
    *world = new World();

    TextureMaterial * m1 = new TextureMaterial();
    m1->setColorImage(img_w[0], img_h[0], img_chns[0], array_of_images[0]);

    TextureMaterial * m2 = new TextureMaterial();
    m2->setColorImage(img_w[1], img_h[1], img_chns[1], array_of_images[1]);

    Vector3 color(0.3f, 0.8f, 0.3f);
    Vector3 center(-2.0, 0.0, 0.0);
    float r = 0.5f;
    Sphere * s = new Sphere(center, r, color);

    (*world)->addVisibleObject(s);

    Vector3 color5(1.0f, 0.0f, 0.1f);
    Vector3 center2(0.5, 0.0, 0.0);
    float r2 = 1.5f;
    Sphere * s2 = new Sphere(center2, r2, color5);
    s2->setMaterial(*m1);
    (*world)->addVisibleObject(s2);

    float beam_angle = 10.0;
    float falloff_angle = 30.0;
    beam_angle = beam_angle * PI / 180.0;
    falloff_angle = falloff_angle * PI / 180.0;
    Vector3 spotlightpos(-3.0, 3.0, 0.0f);
    Vector3 spotlightdir = - spotlightpos;
    SpotLight * spotlight = new SpotLight(spotlightpos, spotlightdir, beam_angle, falloff_angle);
    (*world)->addLight(spotlight);

    Vector3 spotlightpos2(-4.0f, 0.0, 0.0);
    Vector3 spotlightdir2 = - spotlightpos2;
    SpotLight * spotlight2 = new SpotLight(spotlightpos2, spotlightdir2, beam_angle, falloff_angle);
    // (*world)->addLight(spotlight2);

    Vector3 area_light_pos(-4.0, 2.0, 0);
    Vector3 area_light_dir = - area_light_pos;
    Vector3 area_light_up(0.0, 1.0, 0.0);
    AreaLight * areaLigth = new AreaLight(area_light_pos, area_light_dir, area_light_up, 0.1, 0.1);
    // (*world)->addLight(areaLigth);


    Vector3 color2(0.5f, 1.0f, 0.25f);
    Vector3 point(0.0, -2.5, 0.0);
    Vector3 normal(0, 1.0, 0.0);
    Plane * p = new Plane(normal, point, color2);
    p->setMaterial(*m2);
    (*world)->addVisibleObject(p);

    Vector3 color3(0.1f, 0.2f, 0.8f);
    Vector3 point2(2.5, 0.0, 0.0);
    Vector3 normal2(-1.0, 0.2, 0.2f);
    Plane * p2 = new Plane(normal2, point2, color3);
    p2->setMaterial(*m2);
    (*world)->addVisibleObject(p2);

    Vector3 positioncam(-3.0, 0.0, 4.0);
    Vector3 lookat(0.0f, 0.0f, 0.0f);
    Vector3 direction = lookat - positioncam;
    Vector3 updir(0.0, 1.0, 0.0);
    float aspect_ratio = (float(w))/(float(h));
    float distance_from_screen = 1.0;
    Camera * cam = new Camera(positioncam, direction, updir, aspect_ratio, 1.0, distance_from_screen);
    (*world)->setCamera(*cam);
}

/*  Function: addWorldToEngine
//
//	The function initializes the RenderEngine
//  An already initialized World object is passed to the RenderEngine
//
//	Parameters:
//      int w: Width of the rendered image
//      int h: Height of the rendered image		
//		RenderEngine ** r_engine: Pointer to a pointer to the RenderEngine object
//      World ** world: Pointer to a pointer 	
// 
//	Return:
//		void
*/
__global__
void addWorldToEngine(int w, int h, RenderEngine ** r_engine, World ** world, int samples) {
    *r_engine = new RenderEngine(w, h, **world);
    (* r_engine)->setAntiAliasing(samples);
}



/*  Function: Parallelize Render for each pixels
//
//	The kernel CUDA function implements the parallel threads for rendering each pixel.
//  The rendered pixels are stored in the frame_buffer array
//
//	Parameters:
//
//		
//		
//	
//	Return:
//		void
*/
__global__
void renderPixels(RenderEngine ** r_engine, Vector3 * frame_buffer, hiprandState * rand_sequence, int w, int h) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int index_ij = j * w + i;

    hiprand_init(1984 + index_ij, 0, 0, &rand_sequence[index_ij]);

    frame_buffer[index_ij] =  (*r_engine)->renderPixelSampling(i, j, rand_sequence[index_ij]);
    #ifdef CUDADEBUG
    printf("End of renderPixels\n");
    printf("framebuffer: i: %d r: %d c: %d\n", index_ij, i, j);
    #endif
}


/*  Function: main
//
//	Parses the argument list. Initializes the relevant objects and starts rendering.
//
//	Parameters:
//
//		int argc: Number of arguments
//		char *argv[]: List of the arguments
//	
//	Return:
//		int: 0 if successful
*/
int main(int argc, char *argv[]) {

    // Loading images for textures
    int n_imgs = 2;
    unsigned char * host_imgs[n_imgs];
    int img_w[n_imgs], img_h[n_imgs], img_chns[n_imgs];

    // -------- Load Images Here ------- //
    host_imgs[0] = stbi_load("textures/universe.jpg", &img_w[0], &img_h[0], &img_chns[0], 0);

    host_imgs[1] = stbi_load("textures/wall.jpg", &img_w[1], &img_h[1], &img_chns[1], 0);


    // Allocating devices memory to the images on the device
    unsigned char * temp_array[n_imgs];
    unsigned char ** array_of_images = 0; // Pointer to be allocated device memory
    int * img_w_d;
    int * img_h_d;
    int * img_chns_d;

    gpuErrchk(hipMalloc(&img_w_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_w_d, img_w, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&img_h_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_h_d, img_h, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&img_chns_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_chns_d, img_chns, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    if(array_of_images == 0) {
        gpuErrchk(hipMalloc(&array_of_images,  sizeof(unsigned char*)  * n_imgs));
    }
    for(int i = 0; i < n_imgs; i++) {
        gpuErrchk(hipMalloc(&temp_array[i], img_w[i] * img_h[i] * img_chns[i] * sizeof(unsigned char)));
        gpuErrchk(hipMemcpy(&(array_of_images[i]), &(temp_array[i]), sizeof(unsigned char *), hipMemcpyHostToDevice));//copy child pointer to device
        gpuErrchk(hipMemcpy(temp_array[i], host_imgs[i], img_w[i] * img_h[i] * img_chns[i] * sizeof(unsigned char), hipMemcpyHostToDevice)); // copy image to device
    }
     

    // Creating the required arrays for starting the rendering sequence
    int wid_cuda = 1200, hgt_cuda = 800;

    int samples = 32;

    Vector3 * frame_buffer_cuda;
    gpuErrchk(hipMallocManaged(&frame_buffer_cuda, wid_cuda * hgt_cuda * sizeof(Vector3)));

    hiprandState * rand_sequence;
    gpuErrchk(hipMallocManaged(&rand_sequence, wid_cuda * hgt_cuda * sizeof(hiprandState)));

    // Double Pointer: Done so that memory could be directly allocated to the object 
    // with the call of new constructor inside the global function.
    World ** world_cuda;
    gpuErrchk(hipMallocManaged(&world_cuda, sizeof(World *)));

    RenderEngine ** r_engine_cuda;
    gpuErrchk(hipMallocManaged(&r_engine_cuda, sizeof(RenderEngine *)));

    initializeWorld<<<1, 1>>>(world_cuda, wid_cuda, hgt_cuda, array_of_images, img_w_d, img_h_d, img_chns_d, n_imgs);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    addWorldToEngine<<<1, 1>>>(wid_cuda, hgt_cuda, r_engine_cuda, world_cuda, samples);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    const int block_size_side = 16;
    const dim3 block_size(block_size_side, block_size_side);
    const int grid_size_hgt = (hgt_cuda + block_size_side - 1)/block_size_side;
    const int grid_size_wid = (wid_cuda + block_size_side - 1)/block_size_side;
    const dim3 grid_size(grid_size_wid, grid_size_hgt);

    #ifdef CUDADEBUG
    std::cout<<"Grid Sizes: "<<grid_size_hgt<<" "<<grid_size_wid<<std::endl;
    std::cout<<"Block Sizes: "<<block_size_side<<" "<<block_size_side<<std::endl;
    #endif

    renderPixels<<<grid_size, block_size>>>(r_engine_cuda, frame_buffer_cuda, rand_sequence, wid_cuda, hgt_cuda);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    makeImage(frame_buffer_cuda, wid_cuda, hgt_cuda);

    return 0;
}