#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------------------------------------
// 
// 	File name: main.cu
//	Created By: Haard Panchal
//	Create Date: 03/11/2020
//
//	Description:
//		Main file for the Ray Tracing project. The file implements the parallel CUDA algorithm.
//      You can also use it to create the world that would be used to render the final result.
//
//	History:
//		03/10/19: H. Panchal Created the file
//
//  Declaration:
//      N/A
//
// ----------------------------------------------------------------------------------------------------

// #define MESHDEBUG
// #define MATERIALDEBUG
// #define AREALIGHTDEBUG
// #define SHADOWDEBUG
// #define CUDADEBUG
// #define RENDERDEBUG
// #define INITDEBUG

#define ACTUALRENDER

#include <iostream>
#include <math.h>
#include <hiprand/hiprand_kernel.h>

#define STB_IMAGE_IMPLEMENTATION
#include "libs/stb_image.h"

#include "Vector3.h"
#include "Ray.h"

#include "Camera.h"
#include "World.h"

#include "Sphere.h"
#include "Plane.h"
#include "TriangularMesh.h"
#include "PointLight.h"
#include "DirectionalLight.h"
#include "SpotLight.h"
#include "AreaLight.h"

#include "Material.h"
#include "TextureMaterial.h"

#include "RenderEngine.h"


/*  Function: initializeEngine
//
//  The function adds different objects to World.
//  All the object must be initialized onto the heap.
//
//	Parameters:
//      World ** world: A pointer to a pointer to a world object
//		int w: The width of the resulting image
//		int h: The height of the resulting image
//	
//	Return:
//		void
*/
__global__
void initializeWorld(World ** world, int w, int h, unsigned char ** array_of_images, int * img_w, int * img_h, int * img_chns, int n_imgs) {
    *world = new World();

    TextureMaterial * m1 = new TextureMaterial();
    m1->setColorImage(img_w[0], img_h[0], img_chns[0], array_of_images[0]);

    TextureMaterial * m2 = new TextureMaterial();
    m2->setColorImage(img_w[1], img_h[1], img_chns[1], array_of_images[1]);

    TextureMaterial * m3 = new TextureMaterial();
    m3->setColorImage(img_w[2], img_h[2], img_chns[2], array_of_images[2]);

    Vector3 color(0.3f, 0.8f, 0.3f);
    Vector3 center(-2.0, 0.0, 0.0);
    float r = 0.5f;
    Sphere * s = new Sphere(center, r, color);
    s->setMaterial(*m3);
    // (*world)->addVisibleObject(s);

    Vector3 color5(1.0f, 0.0f, 0.1f);
    Vector3 center2(0.5, 0.0, 0.0);
    float r2 = 1.5f;
    Sphere * s2 = new Sphere(center2, r2, color5);
    s2->setMaterial(*m1);
    // (*world)->addVisibleObject(s2);

    float beam_angle = 10.0;
    float falloff_angle = 180.0;
    beam_angle = beam_angle * PI / 180.0;
    falloff_angle = falloff_angle * PI / 180.0;
    Vector3 spotlightpos(-3.0, 3.0, 0.0f);
    Vector3 spotlightdir = - spotlightpos;
    SpotLight * spotlight = new SpotLight(spotlightpos, spotlightdir, beam_angle, falloff_angle);
    (*world)->addLight(spotlight);

    Vector3 spotlightpos2(1.0f, 3.0, 4.0);
    Vector3 spotlightdir2 = - spotlightpos2;
    SpotLight * spotlight2 = new SpotLight(spotlightpos2, spotlightdir2, beam_angle, falloff_angle);
    (*world)->addLight(spotlight2);

    Vector3 area_light_pos(-4.0, 2.0, 0);
    Vector3 area_light_dir = - area_light_pos;
    Vector3 area_light_up(0.0, 1.0, 0.0);
    AreaLight * areaLigth = new AreaLight(area_light_pos, area_light_dir, area_light_up, 0.1, 0.1);
    // (*world)->addLight(areaLigth);


    Vector3 color2(0.5f, 1.0f, 0.25f);
    Vector3 point(0.0, -2.5, 0.0);
    Vector3 normal(0, 1.0, 0.0);
    Plane * p = new Plane(normal, point, color2);
    p->setMaterial(*m2);
    (*world)->addVisibleObject(p);

    Vector3 color3(0.1f, 0.2f, 0.8f);
    Vector3 point2(4.5, 0.0, 0.0);
    Vector3 normal2(-1.0, 0.2, 0.2f);
    Plane * p2 = new Plane(normal2, point2, color3);
    p2->setMaterial(*m2);
    (*world)->addVisibleObject(p2);

    Vector3 positioncam(-3.0, 2.0, 2.0);
    Vector3 lookat(0.0f, 0.0f, 0.0f);
    Vector3 direction = lookat - positioncam;
    Vector3 updir(0.0, 1.0, 0.0);
    float aspect_ratio = (float(w))/(float(h));
    float distance_from_screen = 1.0;
    Camera * cam = new Camera(positioncam, direction, updir, aspect_ratio, 1.0, distance_from_screen);
    (*world)->setCamera(*cam);
}

/*  Function: addWorldToEngine
//
//	The function initializes the RenderEngine
//  An already initialized World object is passed to the RenderEngine
//
//	Parameters:
//      int w: Width of the rendered image
//      int h: Height of the rendered image		
//		RenderEngine ** r_engine: Pointer to a pointer to the RenderEngine object
//      World ** world: Pointer to a pointer 	
// 
//	Return:
//		void
*/
__global__
void addWorldToEngine(int w, int h, RenderEngine ** r_engine, World ** world, int samples) {
    *r_engine = new RenderEngine(w, h, **world);
    (* r_engine)->setAntiAliasing(samples);
}

__global__
void addMeshToWorld(World ** world, Vector3 * mesh_vertex_data, Vector3 * mesh_normal_data, int no_of_triangles, unsigned char ** array_of_images, int * img_w, int * img_h, int * img_chns, int n_imgs) {
    Vector3 center(0.0f, 0.0f, 0.0f);
    Vector3 color(0.0f, 0.0f, 1.0f);

    #ifdef MESHDEBUG
    for(int i = 0; i < no_of_triangles * 3; i++) {
        printf("i: %d V: %f %f %f\n", i, mesh_vertex_data[i].x(), mesh_vertex_data[i].y(), mesh_vertex_data[i].z());
    }
    for(int i = 0; i < no_of_triangles * 3; i++) {
        printf("i: %d N: %f %f %f\n", i, mesh_normal_data[i].x(), mesh_normal_data[i].y(), mesh_normal_data[i].z());
    }
    #endif

    TextureMaterial * m1 = new TextureMaterial();
    m1->setColorImage(img_w[0], img_h[0], img_chns[0], array_of_images[0]);

    TriangularMesh * t_mesh = new TriangularMesh(center, color, mesh_vertex_data, mesh_normal_data, no_of_triangles);
    t_mesh->setMaterial(*m1);
    (*world)->addVisibleObject(t_mesh);
}


/*  Function: Parallelize Render for each pixels
//
//	The kernel CUDA function implements the parallel threads for rendering each pixel.
//  The rendered pixels are stored in the frame_buffer array
//
//	Parameters:
//
//		
//		
//	
//	Return:
//		void
*/
__global__
void renderPixels(RenderEngine ** r_engine, Vector3 * frame_buffer, hiprandState * rand_sequence, int w, int h) {
    const int i = blockIdx.x * blockDim.x + threadIdx.x;
    const int j = blockIdx.y * blockDim.y + threadIdx.y;
    const int index_ij = j * w + i;

    hiprand_init(1984 + index_ij, 0, 0, &rand_sequence[index_ij]);

    frame_buffer[index_ij] =  (*r_engine)->renderPixelSampling(i, j, rand_sequence[index_ij]);
    #ifdef CUDADEBUG
    printf("End of renderPixels\n");
    printf("framebuffer: i: %d r: %d c: %d\n", index_ij, i, j);
    #endif
}


/*  Function: main
//
//	Parses the argument list. Initializes the relevant objects and starts rendering.
//
//	Parameters:
//
//		int argc: Number of arguments
//		char *argv[]: List of the arguments
//	
//	Return:
//		int: 0 if successful
*/
int main(int argc, char *argv[]) {

    // Loading images for textures
    int n_imgs = 3;
    unsigned char * host_imgs[n_imgs];
    int img_w[n_imgs], img_h[n_imgs], img_chns[n_imgs];

    // -------- Load Images Here ------- //
    host_imgs[0] = stbi_load("textures/universe.jpg", &img_w[0], &img_h[0], &img_chns[0], 0);

    host_imgs[1] = stbi_load("textures/wall.jpg", &img_w[1], &img_h[1], &img_chns[1], 0);

    host_imgs[2] = stbi_load("textures/smile.png", &img_w[2], &img_h[2], &img_chns[2], 0);

    #ifdef MATERIALDEBUG
        std::cout<<img_w[2]<<" "<<img_h[2]<<" "<<img_chns[2]<<std::endl;
    #endif

    // Allocating devices memory to the images on the device
    unsigned char * temp_array[n_imgs];
    unsigned char ** array_of_images = 0; // Pointer to be allocated device memory
    int * img_w_d;
    int * img_h_d;
    int * img_chns_d;

    gpuErrchk(hipMalloc(&img_w_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_w_d, img_w, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&img_h_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_h_d, img_h, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc(&img_chns_d, n_imgs * sizeof(int)));
    gpuErrchk(hipMemcpy(img_chns_d, img_chns, n_imgs * sizeof(int), hipMemcpyHostToDevice));

    if(array_of_images == 0) {
        gpuErrchk(hipMalloc(&array_of_images,  sizeof(unsigned char*)  * n_imgs));
    }
    for(int i = 0; i < n_imgs; i++) {
        gpuErrchk(hipMalloc(&temp_array[i], img_w[i] * img_h[i] * img_chns[i] * sizeof(unsigned char)));
        gpuErrchk(hipMemcpy(&(array_of_images[i]), &(temp_array[i]), sizeof(unsigned char *), hipMemcpyHostToDevice));//copy child pointer to device
        gpuErrchk(hipMemcpy(temp_array[i], host_imgs[i], img_w[i] * img_h[i] * img_chns[i] * sizeof(unsigned char), hipMemcpyHostToDevice)); // copy image to device
    }
    
    // Loading Meshes and Normals
    Vector3 ** mesh_vertex_data; 
    Vector3 ** mesh_normal_data;
    gpuErrchk(hipMallocManaged(&mesh_vertex_data, sizeof(Vector3 *)));
    gpuErrchk(hipMallocManaged(&mesh_normal_data, sizeof(Vector3 *)));
    
    std::string obj_file_name = "models/cube.obj";
    int no_of_triangles = loadOBJ(obj_file_name, mesh_vertex_data, mesh_normal_data);


    // Creating the required arrays for starting the rendering sequence
    int wid_cuda = 1200, hgt_cuda = 800;

    int samples = 8;

    Vector3 * frame_buffer_cuda;
    gpuErrchk(hipMallocManaged(&frame_buffer_cuda, wid_cuda * hgt_cuda * sizeof(Vector3)));

    hiprandState * rand_sequence;
    gpuErrchk(hipMallocManaged(&rand_sequence, wid_cuda * hgt_cuda * sizeof(hiprandState)));

    // Double Pointer: Done so that memory could be directly allocated to the object 
    // with the call of new constructor inside the global function.
    World ** world_cuda;
    gpuErrchk(hipMallocManaged(&world_cuda, sizeof(World *)));

    RenderEngine ** r_engine_cuda;
    gpuErrchk(hipMallocManaged(&r_engine_cuda, sizeof(RenderEngine *)));

    initializeWorld<<<1, 1>>>(world_cuda, wid_cuda, hgt_cuda, array_of_images, img_w_d, img_h_d, img_chns_d, n_imgs);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    addMeshToWorld<<<1, 1>>>(world_cuda, *mesh_vertex_data, *mesh_normal_data, no_of_triangles, array_of_images, img_w_d, img_h_d, img_chns_d, n_imgs);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    addWorldToEngine<<<1, 1>>>(wid_cuda, hgt_cuda, r_engine_cuda, world_cuda, samples);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    const int block_size_side = 16;
    const dim3 block_size(block_size_side, block_size_side);
    const int grid_size_hgt = (hgt_cuda + block_size_side - 1)/block_size_side;
    const int grid_size_wid = (wid_cuda + block_size_side - 1)/block_size_side;
    const dim3 grid_size(grid_size_wid, grid_size_hgt);

    #ifdef CUDADEBUG
    std::cout<<"Grid Sizes: "<<grid_size_hgt<<" "<<grid_size_wid<<std::endl;
    std::cout<<"Block Sizes: "<<block_size_side<<" "<<block_size_side<<std::endl;
    #endif

    renderPixels<<<grid_size, block_size>>>(r_engine_cuda, frame_buffer_cuda, rand_sequence, wid_cuda, hgt_cuda);
    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    makeImage(frame_buffer_cuda, wid_cuda, hgt_cuda);

    return 0;
}